﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudawarping.hpp>

using cv::Mat;

int scale_factor = 4;

int main()
{
	Mat frame;
	cv::cuda::GpuMat frame_gpu;

	frame = cv::imread("C:/cat.jpg");

	frame_gpu.upload(frame);
	cv::cuda::resize(frame_gpu, frame_gpu, cv::Size(frame_gpu.cols / scale_factor, frame_gpu.rows / scale_factor), 0, 0, cv::INTER_LINEAR);
	frame_gpu.download(frame);

	if (!frame.empty()) {

		cv::imshow("Frame", frame);
		cv::waitKey(0);
	}
	else {
		std::cout << "Img is empty\n";

	}



    return 0;
}


